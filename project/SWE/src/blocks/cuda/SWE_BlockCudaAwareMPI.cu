/**
 * @file
 * This file is part of SWE.
 *
 * @author Michael Bader, Kaveh Rahnema, Tobias Schnabel
 * @author Sebastian Rettenberger (rettenbs AT in.tum.de, http://www5.in.tum.de/wiki/index.php/Sebastian_Rettenberger,_M.Sc.)
 *
 * @section LICENSE
 *
 * SWE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SWE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SWE.  If not, see <http://www.gnu.org/licenses/>.
 *
 */


#include "blocks/SWE_Block.hh"
#include "SWE_BlockCudaAwareMPI.hh"
#include "SWE_BlockCUDA_kernels.hh"

#include "SWE_WavePropagationBlockCudaAwareMPI.hh"

#include "tools/help.hh"
#include "tools/Logger.hh"

#include <cassert>
#include <cstdlib>
#include <cmath>

//uncomment this to initialize in gpu
//#define INIT

using namespace std;

/*
 * helper function to read CUDA error codes
 * (implementation in swe.cu */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

/*
 * helper function to read CUDA error codes
 * (implementation in swe.cu */
void tryCUDA(hipError_t err, const char *msg)
{
    if( hipSuccess != err)
    {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

SWE_Block* getCudaBlockInstance(float nx, float ny, float dx, float dy) {
  SWE_Block *block = new SWE_WavePropagationBlockCudaAwareMPI(nx, ny, dx, dy);
  return block;
}

/**
 * Constructor: allocate variables for simulation
 *
 * unknowns h,hu,hv,b are defined on grid indices [0,..,nx+1]*[0,..,ny+1]
 * -> computational domain is [1,..,nx]*[1,..,ny]
 * -> plus ghost cell layer
 *
 * flux terms are defined for edges with indices [0,..,nx]*[1,..,ny]
 * or [1,..,nx]*[0,..,ny] (for horizontal/vertical edges)
 * Flux term with index (i,j) is located on the edge between
 * cells with index (i,j) and (i+1,j) or (i,j+1)
 *
 * bathymetry source terms are defined for cells with indices [1,..,nx]*[1,..,ny]
 *
 *
 * @param i_cudaDevice ID of the CUDA-device, which should be used.
 */
SWE_BlockCudaAwareMPI::SWE_BlockCudaAwareMPI(
	int l_nx, int l_ny,
	float l_dx, float l_dy)
  : SWE_Block(l_nx, l_ny, l_dx, l_dy)
{
  if (nx % TILE_SIZE != 0) {
	cout << "WARNING: nx not a multiple of TILE_SIZE  -> will lead to crashes!"
	  << endl << flush;
  };
  if (ny % TILE_SIZE != 0) {
	cout << "WARNING: ny not a multiple of TILE_SIZE  -> will lead to crashes!"
	  << endl << flush;
  };
#ifdef PACKING_DATA
  // allocate consecutive memory for 2 columns with three unknowns each
  // (h, hu, hv, excluding b) for copy/ghost layer at bottom/top boundary
  int size = nx + 2;
  //bottomLayer = new float[6 * size];
  hipMallocManaged((void**) &bottomLayer, 6 * size * sizeof(float));
  checkCUDAError("allocate Unified memory for bottom copy/ghost layer");
  bottomGhostLayer = new SWE_Block1D(bottomLayer, bottomLayer + size, bottomLayer + (2 * size), size);
  bottomCopyLayer = new SWE_Block1D(bottomLayer + (3 * size), bottomLayer + (4 * size), bottomLayer + (5 * size), size);

  // same for top boundary:
  //topLayer = new float[6 * size];
  hipMallocManaged((void**) &topLayer, 6 * size * sizeof(float));
  checkCUDAError("allocate Unified memory for top copy/ghost layer");
  topGhostLayer = new SWE_Block1D(topLayer, topLayer + size, topLayer + (2 * size), size);
  topCopyLayer = new SWE_Block1D(topLayer + (3 * size), topLayer + (4 * size), topLayer + (5 * size), size);
#endif

}

void SWE_BlockCudaAwareMPI::initScenario(float _offsetX, float _offsetY,
                                 SWE_Scenario &i_scenario,
                                 const bool i_multipleBlocks)
{
  offsetX = _offsetX;
  offsetY = _offsetY;
#if defined(INIT)
    {
      float *hd;
      float *hud;
      float *hvd;
      float *bd;
      hipMallocManaged(&hd, (nx + 2) * (ny + 2) * sizeof(float));
      hipMallocManaged(&hud, (nx + 2) * (ny + 2) * sizeof(float));
      hipMallocManaged(&hvd, (nx + 2) * (ny + 2) * sizeof(float));
      hipMallocManaged(&hvd, (nx + 2) * (ny + 2) * sizeof(float));
      hipMallocManaged(&bd, (nx + 2) * (ny + 2) * sizeof(float));

      for (int i = 1; i <= nx; i++)
      {
        for (int j = 1; j <= ny; j++)
        {
          float x = offsetX + (i - 0.5f) * dx;
          float y = offsetY + (j - 0.5f) * dy;
          hd[(i) * (ny + 2) + (j)] = i_scenario.getWaterHeight(x, y);
          hud[(i) * (ny + 2) + (j)] = i_scenario.getVeloc_u(x, y) * hd[(i) * (ny + 2) + (j)];
          hvd[(i) * (ny + 2) + (j)] = i_scenario.getVeloc_v(x, y) * hd[(i) * (ny + 2) + (j)];
        }
      }

      for (int i = 0; i <= nx + 1; i++)
      {
        for (int j = 0; j <= ny + 1; j++)
        {
          bd[i * (ny + 2) + j] = i_scenario.getBathymetry(offsetX + (i - 0.5f) * dx,
                                                          offsetY + (j - 0.5f) * dy);
        }
      }

      //initialize water height and discharge
      dim3 dimBlock(TILE_SIZE, TILE_SIZE);
      dim3 dimGrid(nx / TILE_SIZE, ny / TILE_SIZE);


      initScenario_cuda<<<dimGrid, dimBlock>>>(h.elemVector(), hu.elemVector(), hv.elemVector(), b.elemVector(), hd, hud, hvd, bd, nx, ny);

    }
#else
    {
      for (int i = 1; i <= nx; i++)
      {
        for (int j = 1; j <= ny; j++)
        {
          float x = offsetX + (i - 0.5f) * dx;
          float y = offsetY + (j - 0.5f) * dy;
          h[i][j] = i_scenario.getWaterHeight(x, y);
          hu[i][j] = i_scenario.getVeloc_u(x, y) * h[i][j];
          hv[i][j] = i_scenario.getVeloc_v(x, y) * h[i][j];
        }
      }

      // initialize bathymetry
      for (int i = 0; i <= nx + 1; i++)
      {
        for (int j = 0; j <= ny + 1; j++)
        {
          b[i][j] = i_scenario.getBathymetry(offsetX + (i - 0.5f) * dx,
                                             offsetY + (j - 0.5f) * dy);
        }
      }
    }
#endif


  // in the case of multiple blocks the calling routine takes care about proper boundary conditions.
  if (i_multipleBlocks == false)
  {
    // obtain boundary conditions for all four edges from scenario
    setBoundaryType(BND_LEFT, i_scenario.getBoundaryType(BND_LEFT));
    setBoundaryType(BND_RIGHT, i_scenario.getBoundaryType(BND_RIGHT));
    setBoundaryType(BND_BOTTOM, i_scenario.getBoundaryType(BND_BOTTOM));
    setBoundaryType(BND_TOP, i_scenario.getBoundaryType(BND_TOP));
  }

  // perform update after external write to variables
  synchAfterWrite();
}

/**
 * Destructor: de-allocate all variables
 */
SWE_BlockCudaAwareMPI::~SWE_BlockCudaAwareMPI() {
#ifdef PACKING_DATA
	hipFree(topLayer);
	hipFree(bottomLayer);
#endif
}

//==================================================================
// methods for simulation
//==================================================================

/**
 * set the values of all ghost cells depending on the specifed
 * boundary conditions
 */
void SWE_BlockCudaAwareMPI::setBoundaryConditions() {
#ifdef DBG
 cout << "Call kernel to compute h in ghost layer corner (for visualisation only) "
      << flush << endl;
#endif

#ifdef DBG
 cout << "Call kernel to compute left/right boundaries " << flush << endl;
#endif
//   synchWaterHeightAfterWrite();
//   synchDischargeAfterWrite();

  if (boundary[BND_LEFT] == PASSIVE || boundary[BND_LEFT] == CONNECT) {}
  else {
     dim3 dimBlock(1,TILE_SIZE);
     dim3 dimGrid(1,ny/TILE_SIZE);
     kernelLeftBoundary<<<dimGrid,dimBlock>>>(
        h.elemVector(),hu.elemVector(),hv.elemVector(),nx,ny,boundary[BND_LEFT]);
  };

  if (boundary[BND_RIGHT] == PASSIVE || boundary[BND_RIGHT] == CONNECT) {}
  else {
     dim3 dimBlock(1,TILE_SIZE);
     dim3 dimGrid(1,ny/TILE_SIZE);
     kernelRightBoundary<<<dimGrid,dimBlock>>>(
        h.elemVector(),hu.elemVector(),hv.elemVector(),nx,ny,boundary[BND_RIGHT]);
  };

#ifdef DBG
  cout << "Call kernel to compute bottom/top boundaries " << flush << endl;
#endif
  if (boundary[BND_BOTTOM] == PASSIVE || boundary[BND_BOTTOM] == CONNECT) {}
  else
  {
      dim3 dimBlock(TILE_SIZE,1);
      dim3 dimGrid(nx/TILE_SIZE,1);
      kernelBottomBoundary<<<dimGrid,dimBlock>>>(
         h.elemVector(),hu.elemVector(),hv.elemVector(),nx,ny,boundary[BND_BOTTOM]);
  }
  if (boundary[BND_TOP] == PASSIVE || boundary[BND_TOP] == CONNECT) {}
  else
  {
      dim3 dimBlock(TILE_SIZE,1);
      dim3 dimGrid(nx/TILE_SIZE,1);
      kernelTopBoundary<<<dimGrid,dimBlock>>>(
         h.elemVector(),hu.elemVector(),hv.elemVector(),nx,ny,boundary[BND_TOP]);
  }

  kernelHdBufferEdges<<<1,1>>>(h.elemVector(), nx, ny);
  kernelHdBufferEdges<<<1,1>>>(hu.elemVector(), nx, ny);
  kernelHdBufferEdges<<<1,1>>>(hv.elemVector(), nx, ny);

}

#ifdef PACKING_DATA
void SWE_BlockCudaAwareMPI::synchCopyLayerBeforeRead()
{
#ifdef DBG
  cout << "Packing data ..." << flush << endl;
#endif

  int size = 3 * (nx + 2);
  // bottom copy layer
  if (boundary[BND_BOTTOM] == PASSIVE || boundary[BND_BOTTOM] == CONNECT)
  {
    dim3 dimBlock(TILE_SIZE, 1);
    dim3 dimGrid(nx / TILE_SIZE, 1);
    kernelBottomCopyLayer<<<dimGrid, dimBlock>>>(
        h.elemVector(), hu.elemVector(), hv.elemVector(), bottomLayer + size, nx, ny);
  };

  // top copy layer
  if (boundary[BND_TOP] == PASSIVE || boundary[BND_TOP] == CONNECT)
  {
    dim3 dimBlock(TILE_SIZE, 1);
    dim3 dimGrid(nx / TILE_SIZE, 1);
    kernelTopCopyLayer<<<dimGrid, dimBlock>>>(
        h.elemVector(), hu.elemVector(), hv.elemVector(), topLayer + size, nx, ny);
  };

	hipDeviceSynchronize();

}

void SWE_BlockCudaAwareMPI::synchGhostLayerAfterWrite()
{
#ifdef DBG
  cout << "Unpacking data ..." << flush << endl;
#endif

  // bottom copy layer
  if (boundary[BND_BOTTOM] == PASSIVE || boundary[BND_BOTTOM] == CONNECT)
  {
    dim3 dimBlock(TILE_SIZE, 1);
    dim3 dimGrid(nx / TILE_SIZE, 1);
    kernelBottomGhostBoundary<<<dimGrid, dimBlock>>>(
        h.elemVector(), hu.elemVector(), hv.elemVector(), bottomLayer, nx, ny);
  };

  // top copy layer
  if (boundary[BND_TOP] == PASSIVE || boundary[BND_TOP] == CONNECT)
  {
    dim3 dimBlock(TILE_SIZE, 1);
    dim3 dimGrid(nx / TILE_SIZE, 1);
    kernelTopGhostBoundary<<<dimGrid, dimBlock>>>(
        h.elemVector(), hu.elemVector(), hv.elemVector(), topLayer, nx, ny);
  };
	hipDeviceSynchronize();

}
#endif
/**
 * register the row or column layer next to a boundary as a "copy layer",
 * from which values will be copied into the ghost layer or a neighbour;
 * @return	a SWE_Block1D object that contains row variables h, hu, and hv
 */
SWE_Block1D* SWE_BlockCudaAwareMPI::registerCopyLayer(BoundaryEdge edge){

  // for TOP and BOTTOM layer, the implementation is identical to that in SWE_Block
  // for LEFT and RIGHT layer, separate layers are used that avoid strided copies
  // when transferring memory between host and device memory
  switch (edge) {
    case BND_LEFT:
      return new SWE_Block1D( h.getColProxy(1), hu.getColProxy(1), hv.getColProxy(1));
    case BND_RIGHT:
      return new SWE_Block1D( h.getColProxy(nx), hu.getColProxy(nx), hv.getColProxy(nx));
#ifdef PACKING_DATA
    case BND_BOTTOM:
      synchCopyLayerBeforeRead();
      return bottomCopyLayer; //PHUONG: Here need to call synchronize so that it has data before first communication
    case BND_TOP:
      synchCopyLayerBeforeRead();
      return topCopyLayer;
#else
    case BND_BOTTOM:
      return new SWE_Block1D( h.getRowProxy(1), hu.getRowProxy(1), hv.getRowProxy(1));
    case BND_TOP:
      return new SWE_Block1D( h.getRowProxy(ny), hu.getRowProxy(ny), hv.getRowProxy(ny));
#endif
  };
  return NULL;
}

/**
 * "grab" the ghost layer at the specific boundary in order to set boundary values
 * in this ghost layer externally.
 * The boundary conditions at the respective ghost layer is set to PASSIVE,
 * such that the grabbing program component is responsible to provide correct
 * values in the ghost layer, for example by receiving data from a remote
 * copy layer via MPI communication.
 * @param	specified edge
 * @return	a SWE_Block1D object that contains row variables h, hu, and hv
 */
SWE_Block1D* SWE_BlockCudaAwareMPI::grabGhostLayer(BoundaryEdge edge){

  // for TOP and BOTTOM layer, the implementation is identical to that in SWE_Block
  // for LEFT and RIGHT layer, separate layers are used that avoid strided copies
  // when transferring memory between host and device memory
  boundary[edge] = PASSIVE;
  switch (edge) {
    case BND_LEFT:
      return new SWE_Block1D( h.getColProxy(0), hu.getColProxy(0), hv.getColProxy(0));
    case BND_RIGHT:
      return new SWE_Block1D( h.getColProxy(nx+1), hu.getColProxy(nx+1), hv.getColProxy(nx+1));
#ifdef PACKING_DATA
    case BND_BOTTOM:
      return bottomGhostLayer;
    case BND_TOP:
      return topGhostLayer;
#else
    case BND_BOTTOM:
      return new SWE_Block1D( h.getRowProxy(0), hu.getRowProxy(0), hv.getRowProxy(0));
    case BND_TOP:
      return new SWE_Block1D( h.getRowProxy(ny+1), hu.getRowProxy(ny+1), hv.getRowProxy(ny+1));
#endif
  };
  return NULL;
}

/**
 * Print some available information about the CUDA devices.
 */
void SWE_BlockCudaAwareMPI::printDeviceInformation()
{
	tools::Logger::logger.printString("Printing device information");

  //! id of the CUDA device.
  int l_deviceId;
  hipGetDevice(&l_deviceId);

  //! total number of CUDA devices on this host.
  int l_deviceCount;
  hipGetDeviceCount(&l_deviceCount);

  //! drive and runtime version
  int l_driverVersion, l_runtimeVersion;
  hipDriverGetVersion(&l_driverVersion);
  hipRuntimeGetVersion(&l_runtimeVersion);

  //! device properties
  hipDeviceProp_t l_deviceProperty;
  hipGetDeviceProperties(&l_deviceProperty, l_deviceId);

  // print information about the current device

  tools::Logger::logger.cout() << "Current CUDA device (relative to host): " << l_deviceId
                     << " ( " << l_deviceCount << " in total)" << std::endl;

  tools::Logger::logger.cout() << "CUDA device properties: "
                     << l_deviceProperty.name << " (name), "
                     << l_driverVersion << "/" << l_runtimeVersion << " (driver/runtime version), "
                     << l_deviceProperty.major << "." << l_deviceProperty.minor << " (compute capability)"
                     << std::endl;
}

void SWE_BlockCudaAwareMPI::init(int i_cudaDevice)
{
	  tools::Logger::logger.setProcessRank(i_cudaDevice);

	  hipSetDevice(i_cudaDevice);

	  // check for a valid CUDA device id
	  #ifndef NDEBUG
	  int l_deviceCount;
	  hipGetDeviceCount(&l_deviceCount);
	  assert( (i_cudaDevice >= 0) && (i_cudaDevice < l_deviceCount) );
	  #endif

	  printDeviceInformation();

	  // Make sure the cuda device is reset at exit
	  atexit( SWE_BlockCudaAwareMPI::finalize );
}

void SWE_BlockCudaAwareMPI::finalize()
{
	// reset the cuda device
	tools::Logger::logger.printString("Resetting the CUDA devices");
	hipDeviceReset();
}
